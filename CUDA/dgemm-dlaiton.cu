
#include <hip/hip_runtime.h>
const char* dgemm_desc = "Dlaiton, best way dgemm.";

#define THREADSPERBLOCK 256


__global__ 
void dlaitonComeback(int n,float *A,float *B,float *C){
	int tid=threadIdx.x+(blockDim.x*blockIdx.x);
	if(tid>=n*n) return;
	int i=tid%n;
	int j=tid/n;
	for(int k=0;k<n;k++){
        C[i+j*n]+=A[i+k*n]*B[k+j*n];        
    }
}


__host__
void square_dgemm(int n, double* A, double* B, double* C){
	float *d_A,*d_B,*d_C; 
	int n2=n*n;
	int size=n*sizeof(float);
	hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
	hipMemcpy(d_C,C,size,hipMemcpyHostToDevice);
	dlaitonComeback<<<(n2+THREADSPERBLOCK-1)/THREADSPERBLOCK,THREADSPERBLOCK>>>(n,d_A,d_B,d_C);
	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}