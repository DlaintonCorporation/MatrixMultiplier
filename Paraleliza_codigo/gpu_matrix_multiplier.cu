#include <stdio.h>
#include <stdlib.h>

#define ARRAY_SIZE 10

int **AlocMat(int linhas, int colunas) {
   int **v;
   if ((linhas < 1) || (colunas < 1)) 
      return NULL;
   v = (int **)calloc(linhas, sizeof(int*));
   for (int i = 0 ; i < linhas ; i++) {
      v[i] = (int *)calloc(colunas, sizeof(int));
   }
   return v;
}

void fillMatriz(int n, int m, int **matriz){
   for(int i=0;i<n;i++)
      for(int j=0;j<m;j++){
         matriz[i][j]=rand() % 10000;
      }
}

int main(int argc, char *argv[])
{
    int **h_a, **h_out;
    int *d_a;

    int i;

    h_a = AlocMat(ARRAY_SIZE, ARRAY_SIZE);
    h_out = (ARRAY_SIZE, ARRAY_SIZE);

    hipMalloc(&d_a, sz);
}
